#include "hip/hip_runtime.h"
#include "Cost.cuh"
//添加cuda库
#include "hip/hip_runtime.h"
#include ""


Cost::Cost(int nodeNum, int dim, Model* model, MetricsTypeEnum metricsType){
    this->nodeNum = nodeNum;
    this->dim = dim;
    this->model = model;
    metrics = Metrics::create(metricsType);
}

Cost::Cost(int nodeNum, int dim, Model* model, Metrics* metrics) {
    this->nodeNum = nodeNum;
    this->dim = dim;
    this->model = model;
    this->metrics = metrics;
}

Cost::~Cost() {
    delete metrics;
}


__global__ void kernelWrapper(Cost* costFunc, double* pars, double* cost, Flow* data) {
    //TODO: prepare data for device, see 
    // https://stackoverflow.com/questions/39006348/accessing-class-data-members-from-within-cuda-kernel-how-to-design-proper-host
    // https://stackoverflow.com/questions/65325842/how-do-i-properly-implement-classes-whose-members-are-called-both-from-host-and?noredirect=1&lq=1
    // FIXME: 在 host创建的对象的虚函数在device上不能调用，因为虚函数表在host上，所以需要在device上创建对象
    costFunc->execute(pars, cost, data); // 这个地方不work，关键原因是Cost的中有很多不在Cuda上的内存，所以不能直接调用
    
}

void Cost::calculate(double** pars, int parNum, Flow* data, double* cost) {
    
    // generate linear array
    double* LPar = new double[N_PAR * dim];
    for (int i = 0; i < N_PAR; i++) {
        for (int j = 0; j < dim; j++) {
            LPar[i * dim + j] = pars[i][j];
        }
    }

    // allocate memory on GPU
    double* d_Par = NULL;
    double* d_cost = NULL;
    Flow* d_data = NULL;
    hipMalloc((void**)&d_cost, N_PAR * sizeof(double));
    hipMalloc((void**)&d_data, dataConfig->flowNum * sizeof(Flow));
    // FIXME: this malloc turns d_Par to NULL
    hipMalloc((void**)&d_Par, N_PAR * dim * sizeof(double));
    if (d_Par == NULL) {
        throw std::runtime_error("Failed to allocate memory on GPU");
    }

    Cost* d_costFunc = prepareForDevice();
    // copy data from CPU to GPU
    hipMemcpy(d_data, data, dataConfig->flowNum * sizeof(Flow), hipMemcpyHostToDevice);
    hipMemcpy(d_Par, LPar, N_PAR * dim * sizeof(double), hipMemcpyHostToDevice);

    kernelWrapper<<<(N_PAR + (THREADS_PER_BLOCK + 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>
    (d_costFunc, d_Par, d_cost, d_data);
    hipDeviceSynchronize();
    hipMemcpy(cost, d_cost, N_PAR * sizeof(double), hipMemcpyDeviceToHost);

    // release memory
    hipFree(d_Par);
    hipFree(d_cost);
    hipFree(d_data);

    d_costFunc->leaveDevice();
    hipFree(d_costFunc);

    delete[] LPar;

}

Cost* RegularCost::prepareForDevice() {
    // copy all Cost Members from CPU to GPU 
    RegularCost* d_costFunc;
    hipMalloc((void**)&d_costFunc, sizeof(RegularCost));
    hipMemcpy(d_costFunc, this, sizeof(RegularCost), hipMemcpyHostToDevice);
    Model* d_model = model->prepareForDevice();
    Metrics* d_metrics = metrics->prepareForDevice();
    hipMemcpy(&(d_costFunc->model), &d_model, sizeof(Model*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_costFunc->metrics), &d_metrics, sizeof(Metrics*), hipMemcpyHostToDevice);
    return d_costFunc;
}

Cost* PCost::prepareForDevice() {
    PCost* d_costFunc;
    hipMalloc((void**)&d_costFunc, sizeof(PCost));

    d_costFunc->model = model->prepareForDevice();
    d_costFunc->metrics = metrics->prepareForDevice();
    return d_costFunc;
}

void Cost::leaveDevice() {
    metrics->leaveDevice();
    hipFree(metrics);
    model->leaveDevice();
    hipFree(model);
}

void Cost::predict(double* pars, Flow* data, int metricsSize, MetricsTypeEnum metricsTypes[], double* cost) {
    double* pred = new double[dataConfig->flowNum];
    int flowNum = nodeNum * (nodeNum - 1) / 2;
    model->pred(0, pars, pred, data);
    Metrics* m;
    for (int i = 0; i < metricsSize; i++) {
        m = Metrics::create(metricsTypes[i]);
        cost[i] = m->calc(data, pred, flowNum);
        delete m;
    }
    delete pred;
}

// 对应的是 gh 代码的 cost
__device__ void RegularCost::execute(double* pars, double *cost, Flow* data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double* pred;
    int flowNum = nodeNum * (nodeNum - 1) / 2;
    hipMalloc((void**)&pred, flowNum * sizeof(double));
    model->pred(index, pars, pred, data);
    cost[index] = metrics->calc(data, pred, flowNum);
    hipFree(pred);
}

Cost* Cost::create(CostTypeEnum costType, int nodeNum, int dim, Model* model, MetricsTypeEnum metricsType) {
    switch (costType) {
    case CostTypeEnum::Regular:
        return new RegularCost(nodeNum, dim, model, metricsType);
    case CostTypeEnum::P:
        return new PCost(nodeNum, dim, model, metricsType);
    default:
        throw std::runtime_error("Unknown cost type");
    }
}

// void Cost::destroy(Cost* cost) {
//     cost->~Cost();
//     hipFree(cost);
// }

__device__ void PCost::execute(double* pars, double* cost, Flow* data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double* pred;
    int flowNum = nodeNum * (nodeNum - 1) / 2;
    hipMalloc((void**)&pred, flowNum * sizeof(double));
    model->pred(index, pars, pred, data);
    cost[index] = metrics->calc(data, pred, flowNum);
    hipFree(pred);
}
