#include "hip/hip_runtime.h"
#include "Cost.cuh"
//添加cuda库
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>

#include "error_caught.cuh"


Cost::Cost(int nodeNum, int dim, int flowNum, Model* model, MetricsTypeEnum metricsType){
    this->_nodeNum = nodeNum;
    this->_flowNum = flowNum;
    this->_dim = dim;
    this->_model = model;
    _metrics = Metrics::create(metricsType);
}

Cost::Cost(int nodeNum, int dim, int flowNum, Model* model, Metrics* metrics) {
    this->_nodeNum = nodeNum;
    this->_flowNum = flowNum;
    this->_dim = dim;
    this->_model = model;
    this->_metrics = metrics;
}

Cost::~Cost() {
    delete _metrics;
}


__global__ void kernelWrapper(GlobalConfig* config, float* pars, float* cost, FlowData* data) {
    // 在 host创建的对象的虚函数在device上不能调用，因为虚函数表在host上，所以需要在device上创建对象
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=inherit#data-members
    // 所有的CUDA对象都需要在这个函数中创建

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    Model* model = Model::create(config->modelType, config->nodeNum, config->dim, config->flowNum);
    Cost* costFunc = Cost::create(config->costType, config->nodeNum, config->dim, config->flowNum, model, config->metricsType);
    for (int i = 0; i < config->flowNum; i++) {
        if (data[i].src > 50000) {
            printf("Wrapper: flow %d is broken in kernel %d\n", i, index);
        }
        checkCudaErrors(hipGetLastError(), index, data, __FILE__, __LINE__);
    }

    costFunc->_execute(pars, cost, data); // 这个地方不work，关键原因是Cost的中有很多不在Cuda上的内存，所以不能直接调用

    delete model;
    delete costFunc;
}

void Cost::calculate(GlobalConfig config, float** pars, int parNum, FlowData* data, float* cost) {
    
    // generate linear array
    float* LPar = new float[N_PAR * _dim];
    for (int i = 0; i < N_PAR; i++) {
        for (int j = 0; j < _dim; j++) {
            LPar[i * _dim + j] = pars[i][j];
        }
    }
    // allocate memory on GPU
    float* d_Par = NULL;
    float* d_cost = NULL;
    FlowData* d_data = NULL;
    GlobalConfig* d_config;

    hipMalloc((void**)&d_cost, N_PAR * sizeof(float));
    hipMalloc((void**)&d_data, _flowNum * 2* sizeof(FlowData));
    hipMalloc((void**)&d_Par, N_PAR * _dim * sizeof(float));
    hipMalloc((void**)&d_config, sizeof(GlobalConfig));
    // copy data from CPU to GPU
    hipMemcpy(d_data, data, _flowNum * sizeof(FlowData), hipMemcpyHostToDevice);
    hipMemcpy(d_Par, LPar, N_PAR * _dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_config, &config, sizeof(GlobalConfig), hipMemcpyHostToDevice);
    int blockNum = (N_PAR + (_THREADS_PER_BLOCK - 1)) / _THREADS_PER_BLOCK;

    kernelWrapper<<<blockNum, _THREADS_PER_BLOCK>>>
    (d_config, d_Par, d_cost, d_data);
    hipDeviceSynchronize();

    //* Debug test
    // kernelWrapper<<<1,1>>> (d_config, d_Par, d_cost, d_data);
    hipMemcpy(cost, d_cost, N_PAR * sizeof(float), hipMemcpyDeviceToHost);

    // release memory
    hipFree(d_Par);
    hipFree(d_cost);
    hipFree(d_data);
    hipFree(d_config);


    delete[] LPar;

}

Cost* RegularCost::prepareForDevice() {
    // copy all Cost Members from CPU to GPU 
    RegularCost* d_costFunc;
    hipMalloc((void**)&d_costFunc, sizeof(RegularCost));
    hipMemcpy(d_costFunc, this, sizeof(RegularCost), hipMemcpyHostToDevice);
    Model* d_model = _model->prepareForDevice();
    Metrics* d_metrics = _metrics->prepareForDevice();
    hipMemcpy(&(d_costFunc->_model), &d_model, sizeof(Model*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_costFunc->_metrics), &d_metrics, sizeof(Metrics*), hipMemcpyHostToDevice);
    return d_costFunc;
}

Cost* PCost::prepareForDevice() {
    PCost* d_costFunc;
    hipMalloc((void**)&d_costFunc, sizeof(PCost));

    d_costFunc->_model = _model->prepareForDevice();
    d_costFunc->_metrics = _metrics->prepareForDevice();
    return d_costFunc;
}

void Cost::leaveDevice() {
    _metrics->leaveDevice();
    hipFree(_metrics);
    _model->leaveDevice();
    hipFree(_model);
}

void Cost::predict(float* pars, FlowData* data, int metricsSize, MetricsTypeEnum metricsTypes[], float* cost) {
    float* pred = new float[_flowNum];
    _model->pred(0, pars, pred, data);
    Metrics* m;
    for (int i = 0; i < metricsSize; i++) {
        m = Metrics::create(metricsTypes[i]);
        cost[i] = m->calc(data, pred, _flowNum);
        delete m;
    }
    delete pred;
}

// 对应的是 gh 代码的 cost
__device__ void RegularCost::_execute(float* pars, float *cost, FlowData* data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float* pred = NULL;
    hipError_t e = hipMalloc((void**)&pred, _flowNum * sizeof(float));
    if (e != hipSuccess) {
        printf("Pred size %d, malloc Failed in Thread %d, Error: %s, %s\n", _flowNum*sizeof(float), index, hipGetErrorName(e), hipGetErrorString(e));
    }
    checkCudaErrors(hipGetLastError(), index, data, __FILE__, __LINE__);
    // printf("pred size: %d\n", _flowNum * sizeof(float));
    for (int i = 0; i < _flowNum; i++) {
        if (data[i].src > _flowNum) {
            printf("execute: flow %d is broken in kernel %d\n", i, index);
        }
    }

    checkCudaErrors(hipGetLastError(), index, data, __FILE__, __LINE__);
    _model->pred(index, pars, pred, data);
    cost[index] = _metrics->calc(data, pred, _flowNum);
    hipFree(pred);
    checkCudaErrors(hipGetLastError(), index, data, __FILE__, __LINE__);
}

__device__ __host__ Cost* Cost::create(CostTypeEnum costType, int nodeNum, int dim, int flowNum, Model* model, MetricsTypeEnum metricsType) {
    switch (costType) {
    case CostTypeEnum::Regular:
        return new RegularCost(nodeNum, dim, flowNum, model, metricsType);
    case CostTypeEnum::P:
        return new PCost(nodeNum, dim, flowNum, model, metricsType);
    default:
        printf("Error: Cost Type Error, return null by default\n");
        return nullptr;
    }
}

// void Cost::destroy(Cost* cost) {
//     cost->~Cost();
//     hipFree(cost);
// }

__device__ void PCost::_execute(float* pars, float* cost, FlowData* data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float* pred;
    hipMalloc((void**)&pred, _flowNum * sizeof(float));
    _model->pred(index, pars, pred, data);
    cost[index] = _metrics->calc(data, pred, _flowNum);
    hipFree(pred);
}
