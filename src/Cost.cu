#include "hip/hip_runtime.h"
#include "Cost.cuh"
//添加cuda库
#include "hip/hip_runtime.h"
#include ""


Cost::Cost(int nodeNum, int dim, Model* model, MetricsTypeEnum metricsType){
    this->nodeNum = nodeNum;
    this->dim = dim;
    this->model = model;
    this->metrics = Metrics::createMetrics(metricsType);
}

Cost::~Cost() {
    delete metrics;
}

void Cost::calculate(Particle* particles, double* cost, Flow* data) {
    
    // generate linear array
    double* LPar = new double[particles->Npar * particles->dim];
    for (int i = 0; i < particles->Npar; i++) {
        for (int j = 0; j < particles->dim; j++) {
            LPar[i * particles->dim + j] = particles->Par[i][j];
        }
    }

    // allocate memory on GPU
    double* d_Par;
    double* d_cost;
    hipMalloc((void**)&d_Par, particles->Npar * dim * sizeof(double));
    hipMalloc((void**)&d_cost, particles->Npar * sizeof(double));
    
    // copy data from CPU to GPU
    hipMemcpy(d_Par, LPar, particles->Npar * dim * sizeof(double), hipMemcpyHostToDevice);

    execute<<<(particles->Npar + (THREADS_PER_BLOCK + 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>
    (d_Par, cost, data);
    hipMemcpy(cost, d_cost, particles->Npar * sizeof(double), hipMemcpyDeviceToHost);

    // release memory
    hipFree(d_Par);
    hipFree(d_cost);
    delete[] LPar;

}

void Cost::predict(double* pars, Flow* data, int metricsSize, MetricsTypeEnum metricsTypes[], double* cost) {
    double* pred;
    int flowNum = nodeNum * (nodeNum - 1) / 2;
    hipMalloc((void**)&pred, flowNum * sizeof(double));
    model->pred(0, pars, pred, data);
    for (int i = 0; i < metricsSize; i++) {
        metrics = Metrics::createMetrics(metricsTypes[i]);
        cost[i] = metrics->calc(data, pred, flowNum);
    }
    hipFree(pred);
}

// 对应的是 gh 代码的 cost
__global__ void RegularCost::execute(double* pars, double *cost, Flow* data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double* pred;
    int flowNum = nodeNum * (nodeNum - 1) / 2;
    hipMalloc((void**)&pred, flowNum * sizeof(double));
    model->pred(index, pars, pred, data);
    cost[index] = metrics->calc(data, pred, flowNum);
    hipFree(pred);
}

RegularCost::RegularCost(int nodeNum, int dim, Model* model, MetricsTypeEnum metricsType) 
    : Cost(nodeNum, dim, model, metricsType) {
}

PCost::PCost(int nodeNum, int dim, Model* model, MetricsTypeEnum metricsType) 
    : Cost(nodeNum, dim, model, metricsType) {
}

__global__ void PCost::execute(double* pars, double* cost, Flow* data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double* pred;
    int flowNum = nodeNum * (nodeNum - 1) / 2;
    hipMalloc((void**)&pred, flowNum * sizeof(double));
    model->pred(index, pars, pred, data);
    cost[index] = metrics->calc(data, pred, flowNum);
    hipFree(pred);
}
