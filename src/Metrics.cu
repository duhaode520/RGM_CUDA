#include "hip/hip_runtime.h"
#include "Metrics.cuh"
#include "hip/hip_runtime.h"
#include ""

Metrics* Metrics::create(MetricsTypeEnum type) {
    switch (type) {
    case MetricsTypeEnum::RMSE:
        return new RMSEMetric();
    case MetricsTypeEnum::R2:
        return new RsquaredMetric();
    default:
        throw std::runtime_error("Unknown metrics type");
    }
}

Metrics* Metrics::prepareForDevice() {
    Metrics* deviceMetrics;
    hipMalloc((void**)&deviceMetrics, sizeof(Metrics));
    hipMemcpy(deviceMetrics, this, sizeof(Metrics), hipMemcpyHostToDevice);
    return deviceMetrics;
}

void Metrics::leaveDevice() {
    // temporarily do nothing
}

// void Metrics::destroy(Metrics* metrics) {
//     metrics->~Metrics();
//     hipFree(metrics);
// }

__device__ __host__ double RMSEMetric::calc(Flow* data, double* pred, int size) {
    double sum = 0;
    for (int i = 0; i < size; i++) {
        sum += (data[i].flow - pred[i]) * (data[i].flow - pred[i]);
    }
    return sqrt(sum / size);
}

__device__ __host__ double RsquaredMetric::calc(Flow* data, double* pred, int size) {
    double mean = 0;
    for (int i = 0; i < size; i++) {
        mean += data[i].flow;
    }
    mean /= size;

    double ss_tot = 0;
    double ss_res = 0;
    for (int i = 0; i < size; i++) {
        ss_tot += (data[i].flow - mean) * (data[i].flow - mean);
        ss_res += (data[i].flow - pred[i]) * (data[i].flow - pred[i]);
    }
    return 1 - ss_res / ss_tot;
}

