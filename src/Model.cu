#include "hip/hip_runtime.h"
#include "Model.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <sstream>

Model* Model::create(ModelTypeEnum type, int nodeNum, int dim) {
    switch (type) {
    case ModelTypeEnum::Reversed_Gravity:
        return new RGM(nodeNum, dim);
    case ModelTypeEnum::Reversed_Gravity_Exp:
        return new RGM_EXP(nodeNum, dim);
    default:
        throw std::runtime_error("Unknown model type");
    }
}

// void Model::destroy(Model* model) {
//     model->~Model();
//     hipFree(model);
// }

RGM::RGM(int nodeNum, int dim) {
    this->nodeNum = nodeNum;
    this->dim = dim;
    flowNum = (nodeNum - 1) * nodeNum / 2;
    hipMallocManaged(&Push, sizeof(double) * nodeNum);
    hipMallocManaged(&Attr, sizeof(double) * nodeNum);
    hipMallocManaged(&beta, sizeof(double));
}

RGM::~RGM() {
    hipFree(Push);
    hipFree(Attr);
    hipFree(beta);
}

__device__ __host__ void RGM::parse(int index, double* pars) {
    for(int c=0;c<nodeNum;c++) {
        Push[c]=pars[index*dim+ c];
        Attr[c]=pars[index*dim+ nodeNum + c];
    }
    *beta = pars[index*dim + dim-1]/BETA_SCALE;
}

__device__ __host__ void RGM::pred(int index, double* pars, double* pred, Flow* data) {
    // 从 particle 的维度中解析出需要的 Push Attr beta
    parse(index, pars);
    // TODO: 这一步其实是可以用 CUDA 2D 的一些手段搞成并行的，但是我懒得学
    for (int i = 0; i < flowNum; i++) {
        int src = data[i].src;
        int dest = data[i].dest;
        double dist = data[i].dist;
        double gtFlow = data[i].flow;
        pred[i] = FLOW_SCALE * Push[src] * Attr[dest] / pow(dist, *beta);
    }
}

std::string RGM::getResult(double* pars) {
    parse(0, pars);
    std::stringstream ss;
    // int extreme = -1;
    for (int i = 0; i < nodeNum; i++) {
        ss << dataConfig->nodeNames[i] << " " << Push[i] << " " << Attr[i] << std::endl;
    }
    ss << "Beta " <<*beta << std::endl;
    return ss.str();
}   

Model* RGM::prepareForDevice() {
    RGM* d_model;
    hipMalloc((void**)&d_model, sizeof(RGM));
    hipMemcpy(d_model, this, sizeof(RGM), hipMemcpyHostToDevice);
    // 这里目前 RGM 有的剩下的几个指针项 Push Attr Beta
    // 是用 hipMallocManaged 分配的，所以不需要再次拷贝
    return d_model;
} 

void RGM::leaveDevice() {
    // temporarily do nothing.
}

RGM_EXP::RGM_EXP(int nodeNum, int dim) : RGM(nodeNum, dim) {
}

__device__ __host__ void RGM_EXP::pred(int index, double* pars, double* pred, Flow* data) {
    // 从 particle 的维度中解析出需要的 Push Attr beta
    parse(index, pars);

    for (int i = 0; i < flowNum; i++) {
        int src = data[i].src;
        int dest = data[i].dest;
        double dist = data[i].dist;
        double gtFlow = data[i].flow;

        // exp形式的距离衰减
        pred[i] = FLOW_SCALE * Push[src] * Attr[dest] / exp(*beta * dist);
    }
}