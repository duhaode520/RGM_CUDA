#include "hip/hip_runtime.h"
#include "Model.cuh"
#include "PSOConfig.h"

#include "hip/hip_runtime.h"
#include ""

#include <sstream>
#include <assert.h>
__device__ void checkCudaErrors(hipError_t err, const char* file, const int line)
{
    if (err != hipSuccess)
    {
        printf("CUDA error at %s:%d code=%d(%s) \n", file, line, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__device__ __host__ Model* Model::create(ModelTypeEnum type, int nodeNum, int dim, int flowNum) {
    switch (type) {
    case ModelTypeEnum::Reversed_Gravity:
        return new RGM(nodeNum, dim, flowNum);
    case ModelTypeEnum::Reversed_Gravity_Exp:
        return new RGM_EXP(nodeNum, dim, flowNum);
    default:
        printf("Unknown model type\n");
        return nullptr;
    }
}

// void Model::destroy(Model* model) {
//     model->~Model();
//     hipFree(model);
// }

RGM::RGM(int nodeNum, int dim, int flowNum) {
    this->_nodeNum = nodeNum;
    this->_dim = dim;
    this->_flowNum = flowNum;
    #if defined(__CUDA_ARCH__) 
        hipMalloc(&_Push, sizeof(float) * nodeNum);
        hipMalloc(&_Attr, sizeof(float) * nodeNum);
        hipMalloc(&_beta, sizeof(float));
    #else
        hipMallocManaged(&_Push, sizeof(float) * nodeNum);
        hipMallocManaged(&_Attr, sizeof(float) * nodeNum);
        hipMallocManaged(&_beta, sizeof(float));
    #endif // 
    
}


RGM::~RGM() {
    hipFree(_Push);
    hipFree(_Attr);
    hipFree(_beta);
}

__device__ __host__ void RGM::_parse(int index, float* pars) {
    for(int c=0;c<_nodeNum;c++) {
        _Push[c]=pars[index*_dim+ c];
        _Attr[c]=pars[index*_dim+ _nodeNum + c];
    }
    *_beta = pars[index*_dim + _dim-1]/BETA_SCALE;
}

__device__ __host__ void RGM::pred(int index, float* pars, float* pred, FlowData* data) {
    // 从 particle 的维度中解析出需要的 Push Attr beta
    _parse(index, pars);
    // TODO: 这一步其实是可以用 CUDA 2D 的一些手段搞成并行的，但是我懒得学
    for (int i = 0; i < _flowNum; i++) {
        if (data[i].src > _flowNum) {
            printf("RGM::pred: flow %d is changed in kernel %d\n", i, index);
        }
        pred[i] = _FLOW_SCALE * _Push[data[i].src] * _Attr[data[i].dest] / powf(data[i].dist, *_beta);
        checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
    }
}

std::string RGM::getResult(float* pars) {
    _parse(0, pars);
    std::stringstream ss;
    // int extreme = -1;
    for (int i = 0; i < _nodeNum; i++) {
        ss << dataConfig->nodeNames[i] << "," << _Push[i] << "," << _Attr[i] << std::endl;
    }
    ss << "Beta:" <<*_beta << std::endl;
    return ss.str();
}   

Model* RGM::prepareForDevice() {
    RGM* d_model;
    hipMalloc((void**)&d_model, sizeof(RGM));
    hipMemcpy(d_model, this, sizeof(RGM), hipMemcpyHostToDevice);
    // 这里目前 RGM 有的剩下的几个指针项 Push Attr Beta
    // 是用 hipMallocManaged 分配的，所以不需要再次拷贝
    return d_model;
} 

void RGM::leaveDevice() {
    // temporarily do nothing.
}


__device__ __host__ void RGM_EXP::pred(int index, float* pars, float* pred, FlowData* data) {
    // 从 particle 的维度中解析出需要的 Push Attr beta
    _parse(index, pars);

    for (int i = 0; i < _flowNum; i++) {
        // exp形式的距离衰减
        pred[i] = _FLOW_SCALE * _Push[data[i].src] * _Attr[data[i].dest] / exp(*_beta * data[i].dist);
    }
}